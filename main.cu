#include "hip/hip_runtime.h"
/* Flockuda

Copyright (C) 2019 Christian Thomas Jacobs

*/

#include <iostream>
#include <hdf5.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "prey.h"
using namespace std;

__global__ void initialise(Prey *p, float *xrandom, float *yrandom, int nprey, double Lx, double Ly);
__global__ void prey_velocity(Prey *p, int nprey, double dt);
__global__ void prey_location(Prey *p, int nprey, double dt);
__global__ void save(Prey *p, int nprey);

int main()
{
    // Domain
    double Lx = 1000.0;
    double Ly = 200.0;

    // Timestepping.
    double t = 0.0;
    double dt = 0.5;
    int it = 0;
    int nt = 5;

    // Prey.
    int nprey = 2048;
    Prey *prey;
    hipMallocManaged(&prey, nprey*sizeof(Prey));

    // Random number generator.
    hiprandGenerator_t generator;
    float *xrandom;
    float *yrandom;
    hipMallocManaged(&xrandom, nprey*sizeof(float));
    hipMallocManaged(&yrandom, nprey*sizeof(float));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, unsigned(time(NULL)));
    hiprandGenerateUniform(generator, xrandom, nprey);
    hiprandGenerateUniform(generator, yrandom, nprey);

    cout << "Flockuda v1.0.0" << endl;

    // Initialise.
    initialise<<<1024, 1024>>>(prey, xrandom, yrandom, nprey, Lx, Ly);
    hipDeviceSynchronize();

    while(it < nt)
    {
        cout << it << "\t" << t << endl;

        // Compute prey velocities.
        prey_velocity<<<1024, 1024>>>(prey, nprey, dt);
        hipDeviceSynchronize();

        cout << prey[0].x[0] << " " << prey[0].x[1] << endl;

        prey_location<<<1024, 1024>>>(prey, nprey, dt);
        hipDeviceSynchronize();

        save<<<1024, 1024>>>(prey, nprey);
        hipDeviceSynchronize();

        // Update time.
        it += 1;
        t += dt;
    }

    // Free unified memory.
    hipFree(prey);
    hiprandDestroyGenerator(generator);
    hipFree(xrandom);
    hipFree(yrandom);

    return 0;
}

__global__ void prey_velocity(Prey *p, int nprey, double dt)
{
    double f[2];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute force terms.
            f[d] = 10.0;

            // Compute velocity using F = ma.
            p[i].v[d] = (1.0/dt)*p[i].vold[d] + (1.0/p[i].m)*(f[d]);
        }
    }

    return;
}

__global__ void prey_location(Prey *p, int nprey, double dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute location solving dx/dt = v.
            p[i].x[d] = p[i].xold[d] + dt*p[i].v[d];
        }
    }

    return;
}

__global__ void save(Prey *p, int nprey)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        p[i].save();
    }

    return;
}

__global__ void initialise(Prey *p, float *xrandom, float *yrandom, int nprey, double Lx, double Ly)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        p[i].initialise(10.0, (double)Lx*xrandom[i], (double)Ly*yrandom[i]);
    }
    return;
}
