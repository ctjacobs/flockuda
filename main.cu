/* Flockuda

Copyright (C) 2019 Christian Thomas Jacobs

*/

#include <iostream>
#include <H5Part.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "predator.h"
#include "prey.h"
using namespace std;


int main()
{
    cout << "Flockuda v1.0.0" << endl;

    // Domain.
    float Lx = 1000.0;
    float Ly = 200.0;

    // Timestepping.
    float t = 0.0;
    float dt = 0.5;
    int it = 0;
    int nt = 1000;

    // Predator.
    Predator *predator;
    hipMallocManaged(&predator, sizeof(Predator));

    // Prey.
    int nprey = 200;
    Prey *prey;
    hipMallocManaged(&prey, nprey*sizeof(Prey));

    // Random number generator.
    hiprandGenerator_t generator;
    float *xrandom;
    float *yrandom;
    hipMallocManaged(&xrandom, nprey*sizeof(float));
    hipMallocManaged(&yrandom, nprey*sizeof(float));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, unsigned(time(NULL)));
    hiprandGenerateUniform(generator, xrandom, nprey);
    hiprandGenerateUniform(generator, yrandom, nprey);

    // Output file streams.
    H5PartFile *output_prey = H5PartOpenFile("prey.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output_prey, nprey);
    H5PartFile *output_predator = H5PartOpenFile("predator.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output_predator, 1);

    // Initialise.
    initialise_prey<<<200, 200>>>(prey, xrandom, yrandom, nprey, Lx, Ly);
    hipDeviceSynchronize();
    initialise_predator(predator);

    write_prey(output_prey, prey, nprey, it);
    write_predator(output_predator, predator, it);

    
    while(it < nt)
    {
        cout << it << "\t" << t << endl;

        // Compute predator velocity.
        


        // Compute prey velocities.
        prey_velocity<<<200, 200>>>(prey, nprey, dt);
        hipDeviceSynchronize();

        cout << prey[0].v[0] << " " << prey[0].v[1] << endl;

        prey_location<<<200, 200>>>(prey, nprey, dt);
        save_prey<<<200, 200>>>(prey, nprey);
        hipDeviceSynchronize();
        save_predator(predator);

        write_prey(output_prey, prey, nprey, it);
        write_predator(output_predator, predator, it);

        // Update time.
        it += 1;
        t += dt;
    }

    // Free unified memory.
    hipFree(predator);
    hipFree(prey);
    hiprandDestroyGenerator(generator);
    hipFree(xrandom);
    hipFree(yrandom);

    // Close output streams.
    H5PartCloseFile(output_prey);
    H5PartCloseFile(output_predator);

    return 0;
}

