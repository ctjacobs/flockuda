/* Flockuda

Copyright (C) 2019 Christian Thomas Jacobs

*/

#include <iostream>
#include <H5Part.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "predator.h"
#include "prey.h"
using namespace std;


int main()
{
    cout << "Flockuda v1.0.0" << endl;

    // Domain.
    float Lx = 1000.0;
    float Ly = 1000.0;

    // Timestepping.
    float t = 0.0;
    float dt = 0.2;
    int it = 0;
    int nt = 1000;

    // Predator.
    Predator *predator;
    hipMallocManaged(&predator, sizeof(Predator));

    // Prey.
    int nprey = 200;
    Prey *prey;
    hipMallocManaged(&prey, nprey*sizeof(Prey));

    // Centre of flock.
    float c[2];

    // Random number generator.
    hiprandGenerator_t generator;
    float *xrandom;
    float *yrandom;
    hipMallocManaged(&xrandom, nprey*sizeof(float));
    hipMallocManaged(&yrandom, nprey*sizeof(float));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, unsigned(time(NULL)));
    hiprandGenerateUniform(generator, xrandom, nprey);
    hiprandGenerateUniform(generator, yrandom, nprey);

    // Output file streams.
    H5PartFile *output_prey = H5PartOpenFile("prey.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output_prey, nprey);
    H5PartFile *output_predator = H5PartOpenFile("predator.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output_predator, 1);

    // Initialise.
    initialise_prey<<<1, nprey>>>(prey, xrandom, yrandom, nprey, Lx, Ly);
    hipDeviceSynchronize();
    initialise_predator(predator);
    hipDeviceSynchronize();

    // Write initial condition.
    write_prey(output_prey, prey, nprey, it);
    write_predator(output_predator, predator, it);

    // Timestepping loop.
    while(it < nt)
    {
        cout << "Iteration " << it << "\t Time: " << t << endl;
        cout << prey[0].v[0] << " " << prey[0].v[1] << endl;
        cout << predator->x[0] << " " << predator->x[1] << endl;

        prey_centre(prey, nprey, c);

        // Compute predator velocity.
        predator_velocity(predator, c, xrandom, dt);
        hipDeviceSynchronize();
        predator_location(predator, dt);
        hipDeviceSynchronize();
        save_predator(predator);
        hipDeviceSynchronize();

        // Compute prey velocities.
        hiprandGenerateUniform(generator, xrandom, nprey);
        hipDeviceSynchronize();
        prey_velocity<<<1, nprey>>>(prey, nprey, predator->x[0], predator->x[1], xrandom, dt);
        hipDeviceSynchronize();
        prey_location<<<1, nprey>>>(prey, nprey, dt);
        hipDeviceSynchronize();
        save_prey<<<1, nprey>>>(prey, nprey);
        hipDeviceSynchronize();

        write_prey(output_prey, prey, nprey, it);
        write_predator(output_predator, predator, it);

        // Update time.
        it += 1;
        t += dt;
    }

    // Free unified memory.
    hipFree(predator);
    hipFree(prey);
    hiprandDestroyGenerator(generator);
    hipFree(xrandom);
    hipFree(yrandom);

    // Close output streams.
    H5PartCloseFile(output_prey);
    H5PartCloseFile(output_predator);

    return 0;
}

