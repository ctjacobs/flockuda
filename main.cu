#include "hip/hip_runtime.h"
/* Flockuda

Copyright (C) 2019 Christian Thomas Jacobs

*/

#include <iostream>
#include <H5Part.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "prey.h"
using namespace std;

__global__ void initialise(Prey *p, float *xrandom, float *yrandom, int nprey, float Lx, float Ly);
__global__ void prey_velocity(Prey *p, int nprey, float dt);
__global__ void prey_location(Prey *p, int nprey, float dt);
__global__ void save(Prey *p, int nprey);
__host__ void write(H5PartFile *output, Prey *p, int nprey, int it);

int main()
{
    cout << "Flockuda v1.0.0" << endl;

    // Domain
    float Lx = 1000.0;
    float Ly = 200.0;

    // Timestepping.
    float t = 0.0;
    float dt = 0.5;
    int it = 0;
    int nt = 5;

    // Prey.
    int nprey = 2048;
    Prey *prey;
    hipMallocManaged(&prey, nprey*sizeof(Prey));

    // Random number generator.
    hiprandGenerator_t generator;
    float *xrandom;
    float *yrandom;
    hipMallocManaged(&xrandom, nprey*sizeof(float));
    hipMallocManaged(&yrandom, nprey*sizeof(float));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, unsigned(time(NULL)));
    hiprandGenerateUniform(generator, xrandom, nprey);
    hiprandGenerateUniform(generator, yrandom, nprey);

    // File.
    H5PartFile *output = H5PartOpenFile("prey.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output, nprey);

    // Initialise.
    initialise<<<1024, 1024>>>(prey, xrandom, yrandom, nprey, Lx, Ly);
    hipDeviceSynchronize();

    write(output, prey, nprey, it);

    while(it < nt)
    {
        cout << it << "\t" << t << endl;

        H5PartSetStep(output, it);

        // Compute prey velocities.
        prey_velocity<<<1024, 1024>>>(prey, nprey, dt);
        hipDeviceSynchronize();

        cout << prey[0].x[0] << " " << prey[0].x[1] << endl;

        prey_location<<<1024, 1024>>>(prey, nprey, dt);
        hipDeviceSynchronize();

        save<<<1024, 1024>>>(prey, nprey);
        hipDeviceSynchronize();

        write(output, prey, nprey, it);

        // Update time.
        it += 1;
        t += dt;
    }

    // Free unified memory.
    hipFree(prey);
    hiprandDestroyGenerator(generator);
    hipFree(xrandom);
    hipFree(yrandom);

    // Close output stream.
    H5PartCloseFile(output);

    return 0;
}

__global__ void prey_velocity(Prey *p, int nprey, float dt)
{
    float f[2];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute force terms.
            f[d] = 100.0;

            // Compute velocity using F = ma.
            p[i].v[d] = (1.0/dt)*p[i].vold[d] + (1.0/p[i].m)*(f[d]);
        }
    }

    return;
}

__global__ void prey_location(Prey *p, int nprey, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute location solving dx/dt = v.
            p[i].x[d] = p[i].xold[d] + dt*p[i].v[d];
        }
    }

    return;
}

__global__ void save(Prey *p, int nprey)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        p[i].save();
    }

    return;
}

__global__ void initialise(Prey *p, float *xrandom, float *yrandom, int nprey, float Lx, float Ly)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        p[i].initialise(10.0, Lx*xrandom[i], Ly*yrandom[i]);
    }
    return;
}

__host__ void write(H5PartFile *output, Prey *p, int nprey, int it)
{
    H5PartSetStep(output, it);

    float x[nprey];
    float y[nprey];
    for(int i=0; i <= nprey; ++i)
    {
        x[i] = p[i].x[0];
        y[i] = p[i].x[1];
    }
    H5PartWriteDataFloat32(output, "x", x);
    H5PartWriteDataFloat32(output, "y", y);
}
