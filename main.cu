/*

Flockuda: A numerical model of predator-prey dynamics based on a Molecular Dynamics approach.

Copyright (C) 2019 Christian Thomas Jacobs

*/

#include <iostream>
#include <H5Part.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "predator.h"
#include "prey.h"
using namespace std;


int main()
{
    // All equations solved within are based on those described by Lee et al. (2006), "Prey-Flock Deformation under a Predator's Attack", Journal of the Korean Physical Society, 48:S236--S240.

    cout << "Flockuda v1.0.0" << endl;

    // Domain specification.
    float Lx = 1000.0;
    float Ly = 1000.0;

    // Timestepping parameters.
    float t = 0.0;
    float dt = 0.2;
    int it = 0;
    int nt = 1000;

    // Predator.
    Predator *predator;
    hipMallocManaged(&predator, sizeof(Predator));

    // Prey.
    int nprey = 200;
    Prey *prey;
    hipMallocManaged(&prey, nprey*sizeof(Prey));

    // Centre of flock.
    float centre[2];

    // Random number generator.
    hiprandGenerator_t generator;
    float *xrandom;
    float *yrandom;
    hipMallocManaged(&xrandom, nprey*sizeof(float));
    hipMallocManaged(&yrandom, nprey*sizeof(float));
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, unsigned(time(NULL)));
    hiprandGenerateUniform(generator, xrandom, nprey);
    hiprandGenerateUniform(generator, yrandom, nprey);

    // Output file streams.
    H5PartFile *output_prey = H5PartOpenFile("prey.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output_prey, nprey);
    H5PartFile *output_predator = H5PartOpenFile("predator.h5part", H5PART_WRITE);
    H5PartSetNumParticles(output_predator, 1);

    // Initialise.
    initialise_prey<<<1, nprey>>>(prey, xrandom, yrandom, nprey, Lx, Ly);
    hipDeviceSynchronize();
    initialise_predator(predator);
    hipDeviceSynchronize();

    // Write initial condition.
    write_prey(output_prey, prey, nprey, it);
    write_predator(output_predator, predator, it);

    // Timestepping loop.
    while(it < nt)
    {
        cout << "Iteration " << it << "\t Time: " << t << endl;        

        // Compute the centre of the flock.
        prey_centre(prey, nprey, centre);

        // Compute predator velocity.
        predator_velocity(predator, centre, xrandom, dt);
        predator_location(predator, dt);
        save_predator(predator);

        // Compute prey velocities.
        prey_velocity<<<1, nprey>>>(prey, nprey, predator->x[0], predator->x[1], dt);
        hipDeviceSynchronize();
        prey_location<<<1, nprey>>>(prey, nprey, dt);
        hipDeviceSynchronize();
        save_prey<<<1, nprey>>>(prey, nprey);
        hipDeviceSynchronize();

        // Write prey and predator positions to file.
        write_prey(output_prey, prey, nprey, it);
        write_predator(output_predator, predator, it);

        // Update time.
        it += 1;
        t += dt;
    }

    // Free unified memory.
    hipFree(predator);
    hipFree(prey);
    hiprandDestroyGenerator(generator);
    hipFree(xrandom);
    hipFree(yrandom);

    // Close output streams.
    H5PartCloseFile(output_prey);
    H5PartCloseFile(output_predator);

    return 0;
}

