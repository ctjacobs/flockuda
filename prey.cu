#include "hip/hip_runtime.h"
#include "prey.h"
#include "forces.h"

__device__ void Prey::initialise(float mass, float x0, float x1)
{   
    // Mass.
    m = mass;

    // Location.
    x[0] = x0;
    x[1] = x1;

    // Velocity.
    v[0] = 0.0;
    v[1] = 0.0;

    save();

    return;
}

__device__ void Prey::save()
{   
    // Save information from the previous timestep.
    xold[0] = x[0];
    xold[1] = x[1];
    vold[0] = v[0];
    vold[1] = v[1];

    return;
}

__global__ void initialise_prey(Prey *p, float *xrandom, float *yrandom, int nprey, float Lx, float Ly)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float mass = 10.0;
    if (i < nprey)
    {
        p[i].initialise(mass, Lx*xrandom[i], Ly*yrandom[i]);
    }
    return;
}

__host__ void write_prey(H5PartFile *output, Prey *p, int nprey, int it)
{
    H5PartSetStep(output, it);

    float x[nprey];
    float y[nprey];
    for(int i=0; i <= nprey; ++i)
    {
        x[i] = p[i].x[0];
        y[i] = p[i].x[1];
    }
    H5PartWriteDataFloat32(output, "PreyX", x);
    H5PartWriteDataFloat32(output, "PreyY", y);
}

__global__ void save_prey(Prey *p, int nprey)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        p[i].save();
    }

    return;
}

__global__ void prey_velocity(Prey *p, int nprey, float dt)
{
    float f[2];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute force terms.
            f[d] = prey_alignment(p, nprey, d) + prey_attraction(p, nprey, d) + prey_repulsion(p, nprey, d) - prey_friction(p, nprey, d);

            // Compute velocity using F = ma.
            p[i].v[d] = dt*(p[i].vold[d] + (1.0/p[i].m)*(f[d]));
        }
    }

    return;
}


__global__ void prey_location(Prey *p, int nprey, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute location solving dx/dt = v.
            p[i].x[d] = p[i].xold[d] + dt*p[i].v[d];
        }
    }

    return;
}

__host__ void centre(Prey *p, int nprey, float *c)
{
    c[0] = 0;
    c[1] = 0;
    for(int i=0; i <= nprey; ++i)
    {
        c[0] += p[i].x[0];
        c[1] += p[i].x[1];
    }
    c[0] /= nprey;
    c[1] /= nprey;
    return;
}
