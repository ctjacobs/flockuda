#include "hip/hip_runtime.h"
/*

Flockuda: A numerical model of predator-prey dynamics based on the Molecular Dynamics approach of Lee et al. (2006).

Copyright (C) 2019 Christian T. Jacobs

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

*/


#include "prey.h"
#include "forces.h"
#include <stdio.h>

__host__ void Prey::initialise(float mass, float x0, float x1)
{   
    // Mass.
    m = mass;

    // Location.
    x[0] = x0;
    x[1] = x1;

    // Velocity.
    v[0] = 0.0;
    v[1] = 0.0;

    save();

    return;
}

__host__ void Prey::save()
{   
    // Save information from the previous timestep.
    xold[0] = x[0];
    xold[1] = x[1];
    vold[0] = v[0];
    vold[1] = v[1];

    return;
}

__host__ void initialise_prey(Prey *p, float *xrandom, float *yrandom, int nprey, float Lx, float Ly)
{

    float mass = 1.0;

    // Initialise prey at random locations throughout the domain.
    for(int i=0; i < nprey; ++i)
    {
        p[i].initialise(mass, Lx*xrandom[i], Ly*yrandom[i]);
    }
    return;
}

__host__ void write_prey(H5PartFile *output, Prey *p, int nprey, int it)
{
    // Record the timestep.
    H5PartSetStep(output, it);

    // Collect all prey location data into X and Y arrays.
    float x[nprey];
    float y[nprey];
    float z[nprey];
    for(int i=0; i < nprey; ++i)
    {
        x[i] = p[i].x[0];
        y[i] = p[i].x[1];
        z[i] = 0;
    }

    // Write to .h5part file.
    H5PartWriteDataFloat32(output, "PreyX", x);
    H5PartWriteDataFloat32(output, "PreyY", y);
    H5PartWriteDataFloat32(output, "PreyZ", z);
}

__host__ void save_prey(Prey *p, int nprey)
{
    for(int i=0; i < nprey; ++i)
    {
        p[i].save();
    }

    return;
}

__global__ void prey_velocity(Prey *p, int nprey, float xp0, float xp1, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float f[2];  // The force acting on the prey.

    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute force terms.
            f[d] = prey_alignment(p, nprey, d) + prey_attraction(p, nprey, d) + prey_repulsion(p, nprey, d) - prey_friction(p, nprey, d) - prey_avoid(p, nprey, xp0, xp1, d);

            // Compute velocity using F = ma as per Equation 1 of Lee et al. (2006).
            p[i].v[d] = p[i].vold[d] + dt*(f[d]/p[i].m);
        }
    }

    return;
}

__global__ void prey_location(Prey *p, int nprey, float dt, float Lx, float Ly)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nprey)
    {
        for(int d=0; d<2; ++d)
        {
            // Compute location solving dx/dt = v as per Equation 1 of Lee et al. (2006).
            p[i].x[d] = p[i].xold[d] + dt*p[i].v[d];

            // Apply periodic boundary condition.
            if(d == 0 && p[i].x[d] > Lx)
            {
                p[i].x[d] -= Lx;
                p[i].v[d] = 0;
            }
            else
            {
                if (d == 0 && p[i].x[d] < 0)
                {
                    p[i].x[d] += Lx;
                    p[i].v[d] = 0;
                }
                else
                {
                    if (d == 1 && p[i].x[d] > Ly)
                    {
                        p[i].x[d] -= Ly;
                        p[i].v[d] = 0;
                    }
                    else
                    {
                        if (d == 1 && p[i].x[d] < 0)
                        {
                            p[i].x[d] += Ly;
                            p[i].v[d] = 0;
                        }
                    }
                }
            }
        }
    }
    return;
}

__host__ void prey_centre(Prey *p, int nprey, float *centre)
{
    // The centre of the prey flock based on computing the average of all prey locations.
    centre[0] = 0;
    centre[1] = 0;
    for(int i=0; i<nprey; ++i)
    {
        centre[0] += p[i].xold[0];
        centre[1] += p[i].xold[1];
    }
    centre[0] /= nprey;
    centre[1] /= nprey;
    return;
}
